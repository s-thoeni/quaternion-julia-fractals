
#include <hip/hip_runtime.h>
/**
 * @file pctdemo_processMandelbrotElement.cu
 * 
 * CUDA code to calculate the Mandelbrot Set on a GPU.
 * 
 * Copyright 2011 The MathWorks, Inc.
 */

/** Work out which piece of the global array this thread should operate on */ 
__device__ size_t calculateGlobalIndex() {
    // Which block are we?
    size_t const globalBlockIndex = blockIdx.x + blockIdx.y * gridDim.x;
    // Which thread are we within the block?
    size_t const localThreadIdx = threadIdx.x + blockDim.x * threadIdx.y;
    // How big is each block?
    size_t const threadsPerBlock = blockDim.x*blockDim.y;
    // Which thread are we overall?
    return localThreadIdx + globalBlockIndex*threadsPerBlock;

}

/** The actual Mandelbrot algorithm for a single location */ 
__device__ unsigned int doIterations( double const xPart0, 
                                      double const yPart0,
                                      double const zPart0,
                                      double const wPart0,
                                      double const cx,
                                      double const cy,
                                      double const cz,
                                      double const cw,
                                      unsigned int const maxIters ) {
    // Initialise: z = z0
    double xPart = xPart0;
    double yPart = yPart0;
    double zPart = zPart0;
    double wPart = wPart0;
    unsigned int count = 0;
    // Loop until escape
    while ( ( count <= maxIters )
            && ((xPart*xPart + yPart*yPart + zPart*zPart + wPart*wPart) <= 16.0) ) {
        ++count;
        // Update: z = z*z + z0;
        double const oldXPart = xPart;
        double const oldYPart = yPart;
        double const oldZPart = zPart;
        double const oldWPart = wPart;
        // Quat mult and add constant
        xPart = oldXPart*oldXPart-oldYPart*oldYPart-oldZPart*oldZPart-oldWPart*oldWPart + cx;
        yPart = oldXPart*oldYPart+oldYPart*oldXPart-oldZPart*oldWPart+oldWPart*oldZPart + cy;
        zPart = oldXPart*oldZPart+oldYPart*oldWPart+oldZPart*oldXPart-oldWPart*oldYPart + cz;
        wPart = oldXPart*oldWPart-oldYPart*oldZPart+oldZPart*oldYPart+oldWPart*oldXPart + cw;
        //xPart = xPart*xPart - yPart*yPart + xPart0;
        //yPart = 2.0*oldRealPart*yPart + yPart0;
    }
    return count;
}


/** Main entry point.
 * Works out where the current thread should read/write to global memory
 * and calls doIterations to do the actual work.
 */
__global__ void processMandelbrotElement( 
                      double * out, 
                      const double * x, 
                      const double * y,
                      const double * z,
                      const double * w,
                      const double cx,
                      const double cy,
                      const double cz,
                      const double cw,                      
                      const unsigned int maxIters, 
                      const unsigned int numel ) {
    // Work out which thread we are
    size_t const globalThreadIdx = calculateGlobalIndex();

    // If we're off the end, return now
    if (globalThreadIdx >= numel) {
        return;
    }
    
    // Get our X and Y coords
    double const xPart0 = x[globalThreadIdx];
    double const yPart0 = y[globalThreadIdx];
    double const zPart0 = z[globalThreadIdx];
    double const wPart0 = w[globalThreadIdx];

    // Run the itearations on this location
    unsigned int const count = doIterations( xPart0, yPart0, zPart0, wPart0, cx, cy, cz, cw, maxIters );
    out[globalThreadIdx] =  double( count  );
}
